#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define M 4
#define N 4

__global__ void replaceInnerWithOnesComplement(int *A, int *B, int m, int n) {
    int row = threadIdx.y;
    int col = threadIdx.x;

    if (row < m && col < n) {
        if (row == 0 || row == m - 1 || col == 0 || col == n - 1)
            B[row * n + col] = A[row * n + col];  // Keep border elements same
        else
            B[row * n + col] = ~A[row * n + col]; // 1's complement for inner elements
    }
}

void printMatrix(int *M, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%d ", M[i * cols + j]);
        }
        printf("\n");
    }
}

int main() {
    int size = M * N * sizeof(int);
    int A[M * N] = {1, 2, 3, 4, 6, 5, 8, 3, 2, 4, 10, 1, 9, 1, 2, 5}, B[M * N];

    int *d_A, *d_B;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(N, M);
    replaceInnerWithOnesComplement<<<1, threadsPerBlock>>>(d_A, d_B, M, N);

    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);

    printf("Resultant Matrix B:\n");
    printMatrix(B, M, N);

    hipFree(d_A);
    hipFree(d_B);

    return 0;
}
