#include <stdio.h>
#include <hip/hip_runtime.h>

#define M 2
#define N 4

__global__ void generateString(char *A, int *B, char *output, int *outIndex, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        char ch = A[row * n + col];
        int repeat = B[row * n + col];
        int startIdx = atomicAdd(outIndex, repeat);

        for (int i = 0; i < repeat; i++) {
            output[startIdx + i] = ch;
        }
    }
}

int main() {
    char A[M * N] = {'p', 'C', 'a', 'P', 'e', 'X', 'a', 'M'};
    int B[M * N] = {1, 2, 4, 3, 2, 4, 3, 2};

    int totalSize = 0;
    for (int i = 0; i < M * N; i++) {
        totalSize += B[i];
    }

    char *d_A, *d_output;
    int *d_B, *d_outIndex;
    int outIndex = 0;

    hipMalloc(&d_A, M * N * sizeof(char));
    hipMalloc(&d_B, M * N * sizeof(int));
    hipMalloc(&d_output, totalSize * sizeof(char));
    hipMalloc(&d_outIndex, sizeof(int));

    hipMemcpy(d_A, A, M * N * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, M * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_outIndex, &outIndex, sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(N, M);
    generateString<<<1, threadsPerBlock>>>(d_A, d_B, d_output, d_outIndex, M, N);

    char *output = (char *)malloc(totalSize * sizeof(char));
    hipMemcpy(output, d_output, totalSize * sizeof(char), hipMemcpyDeviceToHost);

    printf("Output String STR: ");
    for (int i = 0; i < totalSize; i++) {
        printf("%c", output[i]);
    }
    printf("\n");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_output);
    hipFree(d_outIndex);
    free(output);

    return 0;
}
