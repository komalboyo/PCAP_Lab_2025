#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define M 3
#define N 3

__global__ void transformMatrix(int *A, int *B, int m, int n) {
    int row = threadIdx.y;
    int col = threadIdx.x;

    if (row < m && col < n) {
        int val = A[row * n + col];
        int power = row + 1;  // Row index determines power
        int result = 1;
        for (int i = 0; i < power; i++) {
            result *= val;
        }
        B[row * n + col] = result;
    }
}

void printMatrix(int *M, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%d ", M[i * cols + j]);
        }
        printf("\n");
    }
}

int main() {
    int size = M * N * sizeof(int);
    int A[M * N] = {1, 2, 3, 4, 5, 6, 7, 8, 9}, B[M * N];

    int *d_A, *d_B;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(N, M);
    transformMatrix<<<1, threadsPerBlock>>>(d_A, d_B, M, N);

    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);

    printf("Resultant Matrix B:\n");
    printMatrix(B, M, N);

    hipFree(d_A);
    hipFree(d_B);

    return 0;
}
