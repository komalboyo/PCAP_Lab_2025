
#include <stdio.h>
#include <hip/hip_runtime.h>
          
#define BLOCK_SIZE 4  
 
__global__ void Scan(int *d_in, int *d_out, int *d_blockSums, int n) {
    __shared__ int temp[BLOCK_SIZE];
 
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + tid;
   
    temp[tid] = (gid < n) ? d_in[gid] : 0;
    __syncthreads();
 
    
    for (int offset = 1; offset <=tid; offset *= 2) {
        int val = temp[tid - offset] ;
        __syncthreads();
        temp[tid] += val;
        __syncthreads();
    }
 
    if (tid == BLOCK_SIZE - 1) {
        d_blockSums[blockIdx.x] = temp[tid];
        printf("block %d : %d\n", blockIdx.x, d_blockSums[blockIdx.x]);
    }
    
 
    if (gid < n) d_out[gid] = temp[tid];
}
__global__ void addBlockOffsets(int *d_out, int *d_blockSums, int n) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
     int i;
    if (blockIdx.x > 0 && gid < n) {
      for(i=0;i<blockIdx.x;i++)
        d_out[gid] += d_blockSums[i];
    }
}
 
int main() {
    printf("Enter length of array: ");
    int N;
    scanf("%d",&N);
    //int h_in[N];
    int * h_in = (int *)malloc(sizeof(int)*N);
    printf("Enter array: ");
    for(int i=0; i<N; i++)
        scanf("%d",&h_in[i]);

    //int h_out[N];
    int * h_out = (int *)malloc(sizeof(int)*N);
 
    int *d_in, *d_out, *d_blockSums;
    int numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
 
    hipMalloc(&d_in, N * sizeof(int));
    int err=hipMalloc(&d_out, N * sizeof(int));
    if(err!=hipSuccess)
    printf("error\n");
    hipMalloc(&d_blockSums, numBlocks * sizeof(int));
 
    hipMemcpy(d_in, h_in, N * sizeof(int), hipMemcpyHostToDevice);
 
    Scan<<<numBlocks, BLOCK_SIZE>>>(d_in, d_out, d_blockSums, N);
 
    addBlockOffsets<<<numBlocks, BLOCK_SIZE>>>(d_out, d_blockSums, N);
 
    hipMemcpy(h_out, d_out, N * sizeof(int), hipMemcpyDeviceToHost);
    printf("Prefix Sum: ");
    for (int i = 0; i < N; i++) 
        printf("%d ", h_out[i]);
    printf("\n");
 
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_blockSums);
    return 0;
}
