// Writ a program in CUDA to add 2 vectors of length N using 
// (a) block number as N
// (b) 1 block N threads


#include<stdio.h>
#include "hip/hip_runtime.h"


__device__ int getGTID(){
	int blockid = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int threadid = blockid * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadid;
}

__global__ void add_a(int *a, int *b, int *c){
	int i = blockIdx.x;
	c[i] = a[i]+b[i];
}

__global__ void add_b(int *a, int *b, int *c){
	int i = threadIdx.x;
	c[i] = a[i]+b[i];
}

int main(){
	int *a, *b, *c;
	int n;
	printf("Enter the size of the vectors: ");
	scanf("%d", &n);
	int s = n*sizeof(int);

	a = (int *)malloc(s);
	b = (int *)malloc(s);
	c = (int *)malloc(s);

	printf("Enter values of array A: ");
	for (int i=0;i<n;i++){
		scanf("%d", &a[i]);
	}
	printf("Enter values of array B: ");
	for (int i=0;i<n;i++){
		scanf("%d", &b[i]);
	}

	int *d_a, *d_b, *d_c;
	hipMalloc((void **)&d_a, s);
	hipMalloc((void **)&d_b, s);
	hipMalloc((void **)&d_c, s);

	hipMemcpy(d_a, a, s, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, s, hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, s, hipMemcpyHostToDevice);

	add_b<<<1, n>>>(d_a, d_b, d_c);
	hipMemcpy(c, d_c, s, hipMemcpyDeviceToHost);

	printf("Result C by method b: ");
	for (int i=0;i<n;i++){
		printf("%d  ", c[i]);
	}
	hipFree(d_c);

	add_a<<<n, 1>>>(d_a, d_b, d_c);
	hipMemcpy(c, d_c, s, hipMemcpyDeviceToHost);

	printf("Result C by method a: ");
	for (int i=0;i<n;i++){
		printf("%d  ", c[i]);
	}
	printf("\n");
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}