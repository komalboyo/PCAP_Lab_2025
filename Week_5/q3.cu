// Process a 1d array containing angles in radians to generate sine of angles in output.


#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void sine_angles(double *a, double *b, int *n){
	int gtid = blockIdx.x * blockDim.x + threadIdx.x;
	if (gtid < *n){
		b[gtid] = sin(a[gtid]);
	}
}

int main(){
	double *a, *b;
	int n;
	printf("Enter the size of the array: ");
	scanf("%d", &n);
	int s = n*sizeof(double);

	a = (double *)malloc(s);
	b = (double *)malloc(s);

	printf("Enter angles in rad: ");
	for (int i=0;i<n;i++){
		scanf("%lf", &a[i]);
	}

	double *d_a, *d_b;
	int *d_n;
	hipMalloc((void **)&d_a, s);
	hipMalloc((void **)&d_b, s);
	hipMalloc((void **)&d_n, sizeof(int));  // treat it as a pointer only
	
	hipMemcpy(d_a, a, s, hipMemcpyHostToDevice);
	hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);

	sine_angles<<<ceil(n/256.0), 256>>>(d_a, d_b, d_n);  // needs to be float for ceil
	hipMemcpy(b, d_b, s, hipMemcpyDeviceToHost);

	printf("Result : \n");
	for (int i=0;i<n;i++){
		printf("sine(%lf) -> %lf  \n", a[i], b[i]);
	}
	printf("\n");
	hipFree(d_a);
	hipFree(d_b);
}