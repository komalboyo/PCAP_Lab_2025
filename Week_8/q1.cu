#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 3 // Define matrix size (N x N for simplicity)

__global__ void addRows(int *A, int *B, int *C, int n) {
    int row = threadIdx.x;
    if (row < n) {
        for (int j = 0; j < n; j++) {
            C[row * n + j] = A[row * n + j] + B[row * n + j];
        }
    }
}

__global__ void addCols(int *A, int *B, int *C, int n) {
    int col = threadIdx.x;
    if (col < n) {
        for (int i = 0; i < n; i++) {
            C[i * n + col] = A[i * n + col] + B[i * n + col];
        }
    }
}

__global__ void addElements(int *A, int *B, int *C, int n) {
    int row = threadIdx.y;
    int col = threadIdx.x;
    if (row < n && col < n) {
        int idx = row * n + col;
        C[idx] = A[idx] + B[idx];
    }
}

void printMatrix(int *M, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d ", M[i * n + j]);
        }
        printf("\n");
    }
}

int main() {
    int size = N * N * sizeof(int);
    int A[N * N], B[N * N], C[N * N];

    // Initialize matrices
    for (int i = 0; i < N * N; i++) {
        A[i] = i;
        B[i] = i * 2;
    }

    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    printf("Original Matrices:\n");
    printMatrix(A, N);
    printf("\n+\n");
    printMatrix(B, N);
    printf("\n=\n");

    // Uncomment the kernel you want to use

    // Case (a): Each row computed by one thread
    // addRows<<<1, N>>>(d_A, d_B, d_C, N);

    // Case (b): Each column computed by one thread
    // addCols<<<1, N>>>(d_A, d_B, d_C, N);

    // Case (c): Each element computed by one thread
    dim3 threadsPerBlock(N, N);
    addElements<<<1, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    printMatrix(C, N);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
